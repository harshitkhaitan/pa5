#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif

#define PI      3.14159256

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------


__global__ void exampleKernel(float *real_image, float *imag_image, int size_x, int size_y)
{
  // Currently does nothing
}

__global__ void pre_compute(float *cos_term, float *sin_term)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;

  float fft_real = 0;
  float fft_imag = 0;

  int tx =  threadIdx.x;

   __syncthreads();
      for(unsigned int n = 0; n < SIZEY; n++)
      {
        float term = -2 * PI * tx * n / SIZEY;
        fft_real = cos(term);
        fft_imag = sin(term);
  	cos_term[n*SIZEY + tx] = fft_real;
  	sin_term[n*SIZEY + tx] = fft_imag;
      }
   __syncthreads();
}

__global__ void pre_compute_i(float *cos_term, float *sin_term)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;

  float fft_real = 0;
  float fft_imag = 0;

  int tx =  threadIdx.x;

   __syncthreads();
      for(unsigned int n = 0; n < SIZEY; n++)
      {
        float term = 2 * PI * tx * n / SIZEY;
        fft_real = cos(term);
        fft_imag = sin(term);
  	cos_term[n*SIZEY + tx] = fft_real;
  	sin_term[n*SIZEY + tx] = fft_imag;
      }
   __syncthreads();
}

__global__ void cpu_fftx_cuda_pre(float *real_image, float *imag_image, int size_x, int size_y, float *cos_term, float *sin_term)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];

  __shared__ float fft_real_s[SIZEY];
  __shared__ float fft_imag_s[SIZEY];
//  __shared__ float realOutBuffer[SIZEY];
//  __shared__ float imagOutBuffer[SIZEY];

//  float fft_real = 0;
//  float fft_imag = 0;
  // Compute the value for this index
  float real_value = 0;
  float imag_value = 0;

//  float real_mul =0 ;
//  float imag_mul =0 ;

  int tx =  threadIdx.x;
  int bx =  blockIdx.x * SIZEY;
  int idx = bx + tx;


     realInBuffer[tx] = real_image[idx];
     imagInBuffer[tx] = imag_image[idx];

    	

      __syncthreads();


      for(unsigned int n = 0; n < SIZEY; n++)
      {
        fft_real_s[tx] = cos_term[n*SIZEY + tx];
        fft_imag_s[tx] = sin_term[n*SIZEY + tx];
//        __syncthreads();

        real_value += (realInBuffer[n] * fft_real_s[tx]) - (imagInBuffer[n] * fft_imag_s[tx]);
        imag_value += (imagInBuffer[n] * fft_real_s[tx]) + (realInBuffer[n] * fft_imag_s[tx]);
//        __syncthreads();
      }


      real_image[idx] = real_value;
      imag_image[idx] = imag_value;
//      __syncthreads();

}

__global__ void cpu_ifftx_cuda_pre(float *real_image, float *imag_image, int size_x, int size_y, float *cos_term, float *sin_term)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];

  __shared__ float fft_real_s[SIZEY];
  __shared__ float fft_imag_s[SIZEY];
//  __shared__ float realOutBuffer[SIZEY];
//  __shared__ float imagOutBuffer[SIZEY];

//  float fft_real = 0;
//  float fft_imag = 0;
  // Compute the value for this index
  float real_value = 0;
  float imag_value = 0;

  int tx =  threadIdx.x;
  int bx =  blockIdx.x * SIZEY;
  int idx = bx + tx;


     realInBuffer[tx] = real_image[idx];
     imagInBuffer[tx] = imag_image[idx];

    	

      __syncthreads();


      for(unsigned int n = 0; n < SIZEY; n++)
      {
 //       float term = -2 * PI * threadIdx.x * n / SIZEY;
 //       fft_real = cos(term);
 //       fft_imag = sin(term);
        fft_real_s[tx] = cos_term[n*SIZEY + tx];
        fft_imag_s[tx] = sin_term[n*SIZEY + tx];
//        __syncthreads();

        real_value += (realInBuffer[n] * fft_real_s[tx]) - (imagInBuffer[n] * fft_imag_s[tx]);
        imag_value += (imagInBuffer[n] * fft_real_s[tx]) + (realInBuffer[n] * fft_imag_s[tx]);
//        __syncthreads();
      }

// Testing 
//	real_value = realInBuffer[tx] * 0.1;
//        imag_value = imagInBuffer[tx]* 5;

//      __syncthreads();

 //      realOutBuffer[threadIdx.x] = real_value;
 //      imagOutBuffer[threadIdx.x] = imag_value;

      real_image[idx] = real_value/SIZEY;
      imag_image[idx] = imag_value/SIZEY;
//      real_image[blockIdx.x*SIZEX + threadIdx.x] = realOutBuffer[threadIdx.x];
//      imag_image[blockIdx.x*SIZEX + threadIdx.x] = imagOutBuffer[threadIdx.x];
//      __syncthreads();

}

__global__ void cpu_ffty_cuda_pre(float *real_image, float *imag_image, int size_x, int size_y, float *cos_term, float *sin_term)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];

  __shared__ float fft_real_s[SIZEY];
  __shared__ float fft_imag_s[SIZEY];
//  __shared__ float realOutBuffer[SIZEY];
//  __shared__ float imagOutBuffer[SIZEY];

//  float fft_real = 0;
//  float fft_imag = 0;
  // Compute the value for this index
  float real_value = 0;
  float imag_value = 0;

  int tx =  threadIdx.x;
  int bx =  blockIdx.x ;
  int idx = bx + tx*SIZEX;


     realInBuffer[tx] = real_image[idx];
     imagInBuffer[tx] = imag_image[idx];

    	

      __syncthreads();


      for(unsigned int n = 0; n < SIZEY; n++)
      {
        fft_real_s[tx] = cos_term[n*SIZEY + tx];
        fft_imag_s[tx] = sin_term[n*SIZEY + tx];

        real_value += (realInBuffer[n] * fft_real_s[tx]) - (imagInBuffer[n] * fft_imag_s[tx]);
        imag_value += (imagInBuffer[n] * fft_real_s[tx]) + (realInBuffer[n] * fft_imag_s[tx]);
//        __syncthreads();
      }


      real_image[idx] = real_value;
      imag_image[idx] = imag_value;
//      __syncthreads();

}

__global__ void cpu_iffty_cuda_pre(float *real_image, float *imag_image, int size_x, int size_y, float *cos_term, float *sin_term)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];

  __shared__ float fft_real_s[SIZEY];
  __shared__ float fft_imag_s[SIZEY];
//  __shared__ float realOutBuffer[SIZEY];
//  __shared__ float imagOutBuffer[SIZEY];

//  float fft_real = 0;
//  float fft_imag = 0;
  // Compute the value for this index
  float real_value = 0;
  float imag_value = 0;

  int tx =  threadIdx.x;
  int bx =  blockIdx.x ;
  int idx = bx + tx*SIZEX;


     realInBuffer[tx] = real_image[idx];
     imagInBuffer[tx] = imag_image[idx];

    	

      __syncthreads();


      for(unsigned int n = 0; n < SIZEY; n++)
      {
        fft_real_s[tx] = cos_term[n*SIZEY + tx];
        fft_imag_s[tx] = sin_term[n*SIZEY + tx];

        real_value += (realInBuffer[n] * fft_real_s[tx]) - (imagInBuffer[n] * fft_imag_s[tx]);
        imag_value += (imagInBuffer[n] * fft_real_s[tx]) + (realInBuffer[n] * fft_imag_s[tx]);
//        __syncthreads();
      }


      real_image[idx] = real_value/SIZEX;
      imag_image[idx] = imag_value/SIZEX;
//      __syncthreads();

}
//__global__ void cpu_fftx_cuda_map(float *real_image, float *imag_image, int size_x, int size_y, float *real_map, float *imag_map)
//{
//  __shared__ float fft_real[SIZEX];
//  __shared__ float fft_imag[SIZEX];
//}

__global__ void cpu_ifftx_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
 // Compute the value for this index
 float real_value = 0;
 float imag_value = 0;
      float fft_real;
      float fft_imag;

  if(threadIdx.x<SIZEY){
      realInBuffer[threadIdx.x] = real_image[blockIdx.x*SIZEX + threadIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[blockIdx.x*SIZEX + threadIdx.x];
      __syncthreads();

      for(unsigned int n = 0; n < SIZEY; n++)
      {
	float term = 2 * PI * threadIdx.x * n / SIZEY;
	fft_real = cos(term);
	fft_imag = sin(term);

	real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
	imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[blockIdx.x*SIZEX + threadIdx.x] = real_value/SIZEY;
      imag_image[blockIdx.x*SIZEX + threadIdx.x] = imag_value/SIZEY;
	
  }	

}

__global__ void cpu_ffty_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
      float fft_real;
      float fft_imag;
      // Compute the value for this index
      float real_value = 0;
      float imag_value = 0;

  if(threadIdx.x<SIZEX){
      realInBuffer[threadIdx.x] = real_image[threadIdx.x*SIZEX + blockIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[threadIdx.x*SIZEX + blockIdx.x];
      __syncthreads();

      for(unsigned int n = 0; n < SIZEX; n++)
      {
        float term = -2 * PI * threadIdx.x * n / SIZEX;
        fft_real = cos(term);
        fft_imag = sin(term);

        real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
        imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[threadIdx.x*SIZEX + blockIdx.x] = real_value;
      imag_image[threadIdx.x*SIZEX + blockIdx.x] = imag_value;
//      __syncthreads();
	
  }	

}

__global__ void cpu_iffty_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
      float fft_real;
      float fft_imag;
      // Compute the value for this index
      float real_value = 0;
      float imag_value = 0;


  if(threadIdx.x<SIZEX){
      realInBuffer[threadIdx.x] = real_image[threadIdx.x*SIZEX + blockIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[threadIdx.x*SIZEX + blockIdx.x];
      __syncthreads();

      for(unsigned int n = 0; n < SIZEX; n++)
      {
        float term = 2 * PI * threadIdx.x * n / SIZEX;
        fft_real = cos(term);
        fft_imag = sin(term);

        real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
        imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[threadIdx.x*SIZEY + blockIdx.x] = real_value/SIZEX;
      imag_image[threadIdx.x*SIZEY + blockIdx.x] = imag_value/SIZEX;
//      __syncthreads();
	
  }	

}
__global__ void cpu_filter_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{
  int eightX = size_x/8;
  int eight7X = size_x - eightX;
  int eightY = size_y/8;
  int eight7Y = size_y - eightY;

  __syncthreads();
  if(!(blockIdx.x < eightX && threadIdx.x < eightY) &&
         !(blockIdx.x < eightX && threadIdx.x >= eight7Y) &&
         !(blockIdx.x >= eight7X && threadIdx.x < eightY) &&
         !(blockIdx.x >= eight7X && threadIdx.x >= eight7Y))
      {
        // Zero out these values
        real_image[threadIdx.x*size_x + blockIdx.x] = 0;
        imag_image[threadIdx.x*size_x + blockIdx.x] = 0;
      }
   __syncthreads();

}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  // Custom measurement
//  hipEvent_t start_me,stop_me;
//  float fftx = 0, ifftx = 0, filter = 0;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));
  

  float *cos_t, *sin_t;
  CUDA_ERROR_CHECK(hipMalloc((void**)&cos_t, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&sin_t, matSize));
//  float *real_m, *imag_m;
//  CUDA_ERROR_CHECK(hipMalloc((void**)&real_m, matSize));
//  CUDA_ERROR_CHECK(hipMalloc((void**)&imag_m, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  //exampleKernel<<<1,128,0,filterStream>>>(device_real,device_imag,size_x,size_y);

//  CUDA_ERROR_CHECK(hipEventCreate(&start_me));
//  CUDA_ERROR_CHECK(hipEventCreate(&stop_me));
//

//  cpu_fftx_cuda_map<<<SIZEY,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y,real_m,imag_m);
//  cpu_fftx_cuda_reduce<<<SIZEY,SIZEX,0,filterStream>>>(device_real,device_imag,size_x,size_y,real_m,imag_m);
dim3 fft_dims;
fft_dims.x = SIZEY;
fft_dims.y = 1;

//  CUDA_ERROR_CHECK(hipEventRecord(start_me,filterStream));
  pre_compute<<<1,SIZEY,0,filterStream>>>(cos_t, sin_t);
  cpu_fftx_cuda_pre<<<fft_dims,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y,cos_t,sin_t);
//  CUDA_ERROR_CHECK(hipEventRecord(stop_me,filterStream));
//  CUDA_ERROR_CHECK(hipEventSynchronize(stop_me));
//  CUDA_ERROR_CHECK(hipEventElapsedTime(&fftx,start_me,stop_me));
//  printf(" Cuda FFTx execution time: %f ms\n", fftx);


  cpu_ffty_cuda_pre<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y,cos_t,sin_t);
  //cpu_ffty_cuda<<<SIZEY,SIZEX,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cpu_filter_cuda<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  pre_compute_i<<<1,SIZEY,0,filterStream>>>(cos_t, sin_t);
  cpu_ifftx_cuda_pre<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y,cos_t,sin_t);
  cpu_iffty_cuda_pre<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y,cos_t,sin_t);


  //cpu_ifftx_cuda<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  //cpu_iffty_cuda<<<SIZEY,SIZEX,0,filterStream>>>(device_real,device_imag,size_x,size_y);

  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));
  CUDA_ERROR_CHECK(hipFree(cos_t));
  CUDA_ERROR_CHECK(hipFree(sin_t));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

