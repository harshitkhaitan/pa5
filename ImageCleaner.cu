#include "hip/hip_runtime.h"
#include "ImageCleaner.h"

#ifndef SIZEX
#error Please define SIZEX.
#endif
#ifndef SIZEY
#error Please define SIZEY.
#endif

#define PI      3.14159256

//----------------------------------------------------------------
// TODO:  CREATE NEW KERNELS HERE.  YOU CAN PLACE YOUR CALLS TO
//        THEM IN THE INDICATED SECTION INSIDE THE 'filterImage'
//        FUNCTION.
//
// BEGIN ADD KERNEL DEFINITIONS
//----------------------------------------------------------------


__global__ void exampleKernel(float *real_image, float *imag_image, int size_x, int size_y)
{
  // Currently does nothing
}

__global__ void cpu_fftx_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
  float fft_real;
  float fft_imag;
  // Compute the value for this index
  float real_value = 0;
  float imag_value = 0;


  if(threadIdx.x<size_y){
      realInBuffer[threadIdx.x] = real_image[blockIdx.x*size_x + threadIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[blockIdx.x*size_x + threadIdx.x];
      __syncthreads();
      for(unsigned int n = 0; n < size_y; n++)
      {
        float term = -2 * PI * threadIdx.x * n / size_y;
        fft_real = cos(term);
        fft_imag = sin(term);

        real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
        imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[blockIdx.x*size_x + threadIdx.x] = real_value;
      imag_image[blockIdx.x*size_x + threadIdx.x] = imag_value;
      __syncthreads();

//     printf("Block Idx %d \n", blockIdx.x);
//      printf("Block DIM %d \n", blockDim.x);
//      printf("Thread ID %d \n", threadIdx.x);
	
  }	

}

__global__ void cpu_ifftx_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
 // Compute the value for this index
 float real_value = 0;
 float imag_value = 0;
      float fft_real;
      float fft_imag;

  if(threadIdx.x<size_y){
      realInBuffer[threadIdx.x] = real_image[blockIdx.x*size_x + threadIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[blockIdx.x*size_x + threadIdx.x];
      __syncthreads();

      for(unsigned int n = 0; n < size_y; n++)
      {
	float term = -2 * PI * threadIdx.x * n / size_y;
	fft_real = cos(term);
	fft_imag = sin(term);

	real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
	imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[blockIdx.x*size_x + threadIdx.x] = real_value/size_y;
      imag_image[blockIdx.x*size_x + threadIdx.x] = imag_value/size_y;
      __syncthreads();

//     printf("Block Idx %d \n", blockIdx.x);
//      printf("Block DIM %d \n", blockDim.x);
//      printf("Thread ID %d \n", threadIdx.x);
	
  }	

}

__global__ void cpu_ffty_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
      float fft_real;
      float fft_imag;
      // Compute the value for this index
      float real_value = 0;
      float imag_value = 0;

  if(threadIdx.x<size_x){
      realInBuffer[threadIdx.x] = real_image[threadIdx.x*size_x + blockIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[threadIdx.x*size_x + blockIdx.x];
      __syncthreads();

      for(unsigned int n = 0; n < size_x; n++)
      {
        float term = -2 * PI * threadIdx.x * n / size_x;
        fft_real = cos(term);
        fft_imag = sin(term);

        real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
        imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[threadIdx.x*size_x + blockIdx.x] = real_value;
      imag_image[threadIdx.x*size_x + blockIdx.x] = imag_value;
      __syncthreads();
	
  }	

}

__global__ void cpu_iffty_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{

//  int BlockIndex = blockIdx.x * blockDim.x;
//  int index = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ float realInBuffer[SIZEY];
  __shared__ float imagInBuffer[SIZEY];
      float fft_real;
      float fft_imag;
      // Compute the value for this index
      float real_value = 0;
      float imag_value = 0;


  if(threadIdx.x<size_x){
      realInBuffer[threadIdx.x] = real_image[threadIdx.x*size_x + blockIdx.x];
      imagInBuffer[threadIdx.x] = imag_image[threadIdx.x*size_x + blockIdx.x];
      __syncthreads();

      for(unsigned int n = 0; n < size_x; n++)
      {
        float term = -2 * PI * threadIdx.x * n / size_x;
        fft_real = cos(term);
        fft_imag = sin(term);

        real_value += (realInBuffer[n] * fft_real) - (imagInBuffer[n] * fft_imag);
        imag_value += (imagInBuffer[n] * fft_real) + (realInBuffer[n] * fft_imag);
      }

      real_image[threadIdx.x*size_y + blockIdx.x] = real_value/size_x;
      imag_image[threadIdx.x*size_y + blockIdx.x] = imag_value/size_x;
      __syncthreads();
	
  }	

}
__global__ void cpu_filter_cuda(float *real_image, float *imag_image, int size_x, int size_y)
{
  int eightX = size_x/8;
  int eight7X = size_x - eightX;
  int eightY = size_y/8;
  int eight7Y = size_y - eightY;

  __syncthreads();
  if(!(blockIdx.x < eightX && threadIdx.x < eightY) &&
         !(blockIdx.x < eightX && threadIdx.x >= eight7Y) &&
         !(blockIdx.x >= eight7X && threadIdx.x < eightY) &&
         !(blockIdx.x >= eight7X && threadIdx.x >= eight7Y))
      {
        // Zero out these values
        real_image[threadIdx.x*size_x + blockIdx.x] = 0;
        imag_image[threadIdx.x*size_x + blockIdx.x] = 0;
      }
   __syncthreads();

}

//----------------------------------------------------------------
// END ADD KERNEL DEFINTIONS
//----------------------------------------------------------------

__host__ float filterImage(float *real_image, float *imag_image, int size_x, int size_y)
{
  // check that the sizes match up
  assert(size_x == SIZEX);
  assert(size_y == SIZEY);

  int matSize = size_x * size_y * sizeof(float);

  // These variables are for timing purposes
  float transferDown = 0, transferUp = 0, execution = 0;
  hipEvent_t start,stop;

  CUDA_ERROR_CHECK(hipEventCreate(&start));
  CUDA_ERROR_CHECK(hipEventCreate(&stop));

  // Create a stream and initialize it
  hipStream_t filterStream;
  CUDA_ERROR_CHECK(hipStreamCreate(&filterStream));

  // Alloc space on the device
  float *device_real, *device_imag;
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_real, matSize));
  CUDA_ERROR_CHECK(hipMalloc((void**)&device_imag, matSize));

  // Start timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));
  
  // Here is where we copy matrices down to the device 
  CUDA_ERROR_CHECK(hipMemcpy(device_real,real_image,matSize,hipMemcpyHostToDevice));
  CUDA_ERROR_CHECK(hipMemcpy(device_imag,imag_image,matSize,hipMemcpyHostToDevice));
  
  // Stop timing for transfer down
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferDown,start,stop));

  // Start timing for the execution
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  //----------------------------------------------------------------
  // TODO: YOU SHOULD PLACE ALL YOUR KERNEL EXECUTIONS
  //        HERE BETWEEN THE CALLS FOR STARTING AND
  //        FINISHING TIMING FOR THE EXECUTION PHASE
  // BEGIN ADD KERNEL CALLS
  //----------------------------------------------------------------

  // This is an example kernel call, you should feel free to create
  // as many kernel calls as you feel are needed for your program
  // Each of the parameters are as follows:
  //    1. Number of thread blocks, can be either int or dim3 (see CUDA manual)
  //    2. Number of threads per thread block, can be either int or dim3 (see CUDA manual)
  //    3. Always should be '0' unless you read the CUDA manual and learn about dynamically allocating shared memory
  //    4. Stream to execute kernel on, should always be 'filterStream'
  //
  // Also note that you pass the pointers to the device memory to the kernel call
  exampleKernel<<<1,128,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cpu_fftx_cuda<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cpu_ffty_cuda<<<SIZEY,SIZEX,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cpu_filter_cuda<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cpu_ifftx_cuda<<<SIZEX,SIZEY,0,filterStream>>>(device_real,device_imag,size_x,size_y);
  cpu_iffty_cuda<<<SIZEY,SIZEX,0,filterStream>>>(device_real,device_imag,size_x,size_y);

  //---------------------------------------------------------------- 
  // END ADD KERNEL CALLS
  //----------------------------------------------------------------

  // Finish timimg for the execution 
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&execution,start,stop));

  // Start timing for the transfer up
  CUDA_ERROR_CHECK(hipEventRecord(start,filterStream));

  // Here is where we copy matrices back from the device 
  CUDA_ERROR_CHECK(hipMemcpy(real_image,device_real,matSize,hipMemcpyDeviceToHost));
  CUDA_ERROR_CHECK(hipMemcpy(imag_image,device_imag,matSize,hipMemcpyDeviceToHost));

  // Finish timing for transfer up
  CUDA_ERROR_CHECK(hipEventRecord(stop,filterStream));
  CUDA_ERROR_CHECK(hipEventSynchronize(stop));
  CUDA_ERROR_CHECK(hipEventElapsedTime(&transferUp,start,stop));

  // Synchronize the stream
  CUDA_ERROR_CHECK(hipStreamSynchronize(filterStream));
  // Destroy the stream
  CUDA_ERROR_CHECK(hipStreamDestroy(filterStream));
  // Destroy the events
  CUDA_ERROR_CHECK(hipEventDestroy(start));
  CUDA_ERROR_CHECK(hipEventDestroy(stop));

  // Free the memory
  CUDA_ERROR_CHECK(hipFree(device_real));
  CUDA_ERROR_CHECK(hipFree(device_imag));

  // Dump some usage statistics
  printf("CUDA IMPLEMENTATION STATISTICS:\n");
  printf("  Host to Device Transfer Time: %f ms\n", transferDown);
  printf("  Kernel(s) Execution Time: %f ms\n", execution);
  printf("  Device to Host Transfer Time: %f ms\n", transferUp);
  float totalTime = transferDown + execution + transferUp;
  printf("  Total CUDA Execution Time: %f ms\n\n", totalTime);
  // Return the total time to transfer and execute
  return totalTime;
}

